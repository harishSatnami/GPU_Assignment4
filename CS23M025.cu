#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <chrono>

using namespace std;

//*******************************************

// Write down the kernels here
__global__ void memset_int(int *arr,int value,int size){
    if(threadIdx.x<size)
    arr[threadIdx.x]=value;
}

__global__ void checktanks(int *gpu_health_write, int *gpu_num_of_tanks_alive){
    if(gpu_health_write[threadIdx.x]<=0){
        atomicSub(&gpu_num_of_tanks_alive[0],1);
    }
}


__global__ void start_current_round(int *gpu_xcoord, int * gpu_ycoord, int *gpu_score, int *gpu_health_write, int *gpu_health_read, int T, int N, int *gpu_shortest_distance, int round_number){
    int source_tank = blockIdx.x;
    int destination_tank = (source_tank + round_number) % T;
    int current_tank = threadIdx.x;
    long long int source_x = gpu_xcoord[source_tank]; 
    long long int source_y = gpu_ycoord[source_tank];
    long long int destination_x = gpu_xcoord[destination_tank]; 
    long long int destination_y = gpu_ycoord[destination_tank];
    long long int current_x = gpu_xcoord[current_tank];
    long long int current_y = gpu_ycoord[current_tank];
    int direction = 1;
    int distance = -1;
    

    if(destination_tank != source_tank && current_tank != source_tank){
        if(destination_y == source_y){
            direction = destination_x > source_x ? 1 : -1;
        }
        else if(source_y > destination_y){
            direction = -1;
        }
    

        if(gpu_health_read[source_tank]>0 && gpu_health_read[current_tank]>0){
            long long int lhs = (destination_x-source_x) * (current_y-source_y);
            long long int rhs = (current_x-source_x) * (destination_y-source_y);

            if(lhs==rhs){
                if((direction == 1 && (source_y < current_y || (source_y == current_y && source_x < current_x))) || (direction == -1 && (source_y > current_y || (current_y == source_y && source_x > current_x)))){
                    distance = abs(current_y - source_y) + abs(current_x - source_x);   
                    atomicMin(&gpu_shortest_distance[source_tank], distance);
                }
            }
        

        }
    }
    __syncthreads();
    if(gpu_shortest_distance[source_tank] == distance) {
        atomicAdd(&gpu_score[source_tank], 1);
        atomicSub(&gpu_health_write[current_tank], 1);
    }
}


//***********************************************


int main(int argc,char **argv)
{
    // Variable declarations
    int M,N,T,H,*xcoord,*ycoord,*score;
    

    FILE *inputfilepointer;
    
    //File Opening for read
    char *inputfilename = argv[1];
    inputfilepointer    = fopen( inputfilename , "r");

    if ( inputfilepointer == NULL )  {
        printf( "input.txt file failed to open." );
        return 0; 
    }

    fscanf( inputfilepointer, "%d", &M );
    fscanf( inputfilepointer, "%d", &N );
    fscanf( inputfilepointer, "%d", &T ); // T is number of Tanks
    fscanf( inputfilepointer, "%d", &H ); // H is the starting Health point of each Tank
	
    // Allocate memory on CPU
    xcoord=(int*)malloc(T * sizeof (int));  // X coordinate of each tank
    ycoord=(int*)malloc(T * sizeof (int));  // Y coordinate of each tank
    score=(int*)malloc(T * sizeof (int));  // Score of each tank (ensure that at the end you have copied back the score calculations on the GPU back to this allocation)

    // Get the Input of Tank coordinates
    for(int i=0;i<T;i++)
    {
      fscanf( inputfilepointer, "%d", &xcoord[i] );
      fscanf( inputfilepointer, "%d", &ycoord[i] );
    }
		

    auto start = chrono::high_resolution_clock::now();

    //*********************************
    // Your Code begins here (Do not change anything in main() above this comment)
    //********************************

    int *gpu_xcoord;
    int *gpu_ycoord;
    int *gpu_score;
    int *gpu_health_write;
    int *gpu_shortest_distance;
    int *gpu_health_read;
    int *gpu_num_of_tanks_alive;

    int num_of_tanks_alive[1] = {T};

    hipMalloc(&gpu_xcoord,sizeof(int) * T);
    hipMalloc(&gpu_ycoord,sizeof(int) * T);
    hipMalloc(&gpu_score,sizeof(int) * T);
    hipMalloc(&gpu_shortest_distance, sizeof(int) * T);
    memset_int<<<1,T>>>(gpu_score, 0, T);
    hipMalloc(&gpu_health_write, sizeof(int) * T);
    hipMalloc(&gpu_health_read, sizeof(int) * T);
    memset_int<<<1,T>>>(gpu_health_write, H, T);
    memset_int<<<1,T>>>(gpu_health_read, H, T);
    hipMemcpy(gpu_xcoord, xcoord, sizeof(int) * T, hipMemcpyHostToDevice);
    hipMemcpy(gpu_ycoord ,ycoord ,sizeof(int) * T, hipMemcpyHostToDevice);
    memset_int<<<1,T>>>(gpu_shortest_distance, INT_MAX, T);
    hipMalloc(&gpu_num_of_tanks_alive, sizeof(int));
    memset_int<<<1,1>>>(gpu_num_of_tanks_alive, T, 1);
    

    for(int round_number = 1; num_of_tanks_alive[0] > 1; round_number++){
        start_current_round<<<T,T>>>(gpu_xcoord, gpu_ycoord, gpu_score, gpu_health_write, gpu_health_read, T, N, gpu_shortest_distance, round_number);
        memset_int<<<1,T>>>(gpu_shortest_distance, INT_MAX, T);
        checktanks<<<1,T>>>(gpu_health_write, gpu_num_of_tanks_alive);
        hipMemcpy(&num_of_tanks_alive[0], gpu_num_of_tanks_alive, sizeof(int), hipMemcpyDeviceToHost);
        memset_int<<<1,1>>>(gpu_num_of_tanks_alive, T, 1);
        hipMemcpy(gpu_health_read,gpu_health_write,sizeof(int) * T, hipMemcpyDeviceToDevice);
        hipDeviceSynchronize();
    }

    hipMemcpy(score, gpu_score, sizeof(int) * T, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();


    //*********************************
    // Your Code ends here (Do not change anything in main() below this comment)
    //********************************

    auto end  = chrono::high_resolution_clock::now();

    chrono::duration<double, std::micro> timeTaken = end-start;

    printf("Execution time : %f\n", timeTaken.count());

    // Output
    char *outputfilename = argv[2];
    char *exectimefilename = argv[3]; 
    FILE *outputfilepointer;
    outputfilepointer = fopen(outputfilename,"w");

    for(int i=0;i<T;i++)
    {
        fprintf( outputfilepointer, "%d\n", score[i]);
    }
    fclose(inputfilepointer);
    fclose(outputfilepointer);

    outputfilepointer = fopen(exectimefilename,"w");
    fprintf(outputfilepointer,"%f", timeTaken.count());
    fclose(outputfilepointer);

    free(xcoord);
    free(ycoord);
    free(score);
    hipDeviceSynchronize();
    return 0;
}